#include "hip/hip_runtime.h"

#include "cuda_utils.cuh"
#include <cstdio>

bool cuda_utils::is_cuda_device_available()
{
    bool CUDA_compatible_device_found = true;
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if ( hipSuccess != hipGetDeviceCount(&deviceCount) || deviceCount == 0 )
    {
        CUDA_compatible_device_found = false;
    }
    return CUDA_compatible_device_found;
}
